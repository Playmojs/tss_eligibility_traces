#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "objects.h"
#include <ctime>
#include <time.h>
#include "run.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "rand.h"

#include "code_objects/synapses_2_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_3_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_1_summed_variable_Igap_post_codeobject.h"
#include "code_objects/neurongroup_1_stateupdater_codeobject.h"
#include "code_objects/neurongroup_2_stateupdater_codeobject.h"
#include "code_objects/neurongroup_stateupdater_codeobject.h"
#include "code_objects/neurongroup_1_spike_thresholder_codeobject.h"
#include "code_objects/after_run_neurongroup_1_spike_thresholder_codeobject.h"
#include "code_objects/neurongroup_2_spike_thresholder_codeobject.h"
#include "code_objects/after_run_neurongroup_2_spike_thresholder_codeobject.h"
#include "code_objects/spikegeneratorgroup_codeobject.h"
#include "code_objects/synapses_2_pre_push_spikes.h"
#include "code_objects/before_run_synapses_2_pre_push_spikes.h"
#include "code_objects/synapses_2_pre_codeobject.h"
#include "code_objects/synapses_3_pre_push_spikes.h"
#include "code_objects/before_run_synapses_3_pre_push_spikes.h"
#include "code_objects/synapses_3_pre_codeobject.h"
#include "code_objects/synapses_pre_push_spikes.h"
#include "code_objects/before_run_synapses_pre_push_spikes.h"
#include "code_objects/synapses_pre_codeobject.h"
#include "code_objects/synapses_1_post_push_spikes.h"
#include "code_objects/before_run_synapses_1_post_push_spikes.h"
#include "code_objects/synapses_1_post_codeobject.h"
#include "code_objects/neurongroup_2_spike_resetter_codeobject.h"


#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime_api.h"




int main(int argc, char **argv)
{
        


    // seed variable set in Python through brian2.seed() calls can use this
    // variable (see device.py CUDAStandaloneDevice.generate_main_source())
    unsigned long long seed;

    //const std::clock_t _start_time = std::clock();

    CUDA_SAFE_CALL(
            hipSetDevice(0)
            );

    hipDeviceProp_t props;
    CUDA_SAFE_CALL(
            hipGetDeviceProperties(&props, 0)
            );
    size_t limit = 128 * 1024 * 1024;
    CUDA_SAFE_CALL(
            hipDeviceSetLimit(hipLimitMallocHeapSize, limit)
            );
    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );

    //const double _run_time2 = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    //printf("INFO: setting cudaDevice stuff took %f seconds\n", _run_time2);

    brian_start();

        


    //const std::clock_t _start_time3 = std::clock();
    {
        using namespace brian;

                
        for(int i=0; i<_num__array_spikegeneratorgroup__spikespace; i++)
        {
            _array_spikegeneratorgroup__spikespace[i] = - 1;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_spikegeneratorgroup__spikespace[current_idx_array_spikegeneratorgroup__spikespace],
                &_array_spikegeneratorgroup__spikespace[0],
                sizeof(_array_spikegeneratorgroup__spikespace[0])*_num__array_spikegeneratorgroup__spikespace,
                hipMemcpyHostToDevice
            )
        );
        _array_spikegeneratorgroup__spikespace[_num__array_spikegeneratorgroup__spikespace - 1] = 0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_spikegeneratorgroup__spikespace[current_idx_array_spikegeneratorgroup__spikespace] + _num__array_spikegeneratorgroup__spikespace - 1,
                                    &_array_spikegeneratorgroup__spikespace[_num__array_spikegeneratorgroup__spikespace - 1],
                                    sizeof(_array_spikegeneratorgroup__spikespace[_num__array_spikegeneratorgroup__spikespace - 1]),
                                    hipMemcpyHostToDevice
                                )
                            );
        for(int i=0; i<_num__array_neurongroup_2__spikespace; i++)
        {
            _array_neurongroup_2__spikespace[i] = - 1;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_neurongroup_2__spikespace[current_idx_array_neurongroup_2__spikespace],
                &_array_neurongroup_2__spikespace[0],
                sizeof(_array_neurongroup_2__spikespace[0])*_num__array_neurongroup_2__spikespace,
                hipMemcpyHostToDevice
            )
        );
        _array_neurongroup_2__spikespace[_num__array_neurongroup_2__spikespace - 1] = 0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_neurongroup_2__spikespace[current_idx_array_neurongroup_2__spikespace] + _num__array_neurongroup_2__spikespace - 1,
                                    &_array_neurongroup_2__spikespace[_num__array_neurongroup_2__spikespace - 1],
                                    sizeof(_array_neurongroup_2__spikespace[_num__array_neurongroup_2__spikespace - 1]),
                                    hipMemcpyHostToDevice
                                )
                            );
        for(int i=0; i<_num__array_neurongroup_1__spikespace; i++)
        {
            _array_neurongroup_1__spikespace[i] = - 1;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_neurongroup_1__spikespace[current_idx_array_neurongroup_1__spikespace],
                &_array_neurongroup_1__spikespace[0],
                sizeof(_array_neurongroup_1__spikespace[0])*_num__array_neurongroup_1__spikespace,
                hipMemcpyHostToDevice
            )
        );
        _array_neurongroup_1__spikespace[_num__array_neurongroup_1__spikespace - 1] = 0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_neurongroup_1__spikespace[current_idx_array_neurongroup_1__spikespace] + _num__array_neurongroup_1__spikespace - 1,
                                    &_array_neurongroup_1__spikespace[_num__array_neurongroup_1__spikespace - 1],
                                    sizeof(_array_neurongroup_1__spikespace[_num__array_neurongroup_1__spikespace - 1]),
                                    hipMemcpyHostToDevice
                                )
                            );
        _array_defaultclock_dt[0] = 0.0001;
        _array_defaultclock_dt[0] = 0.0001;
        _array_defaultclock_dt[0] = 0.0001;
        _dynamic_array_spikegeneratorgroup_spike_number.resize(213084);
        THRUST_CHECK_ERROR(dev_dynamic_array_spikegeneratorgroup_spike_number.resize(213084));
        for(int i=0; i<_num__static_array__dynamic_array_spikegeneratorgroup_spike_number; i++)
        {
            _dynamic_array_spikegeneratorgroup_spike_number[i] = _static_array__dynamic_array_spikegeneratorgroup_spike_number[i];
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_spikegeneratorgroup_spike_number[0]),
                &_dynamic_array_spikegeneratorgroup_spike_number[0],
                sizeof(_dynamic_array_spikegeneratorgroup_spike_number[0])*_dynamic_array_spikegeneratorgroup_spike_number.size(),
                hipMemcpyHostToDevice
            )
        );
        _dynamic_array_spikegeneratorgroup_neuron_index.resize(213084);
        THRUST_CHECK_ERROR(dev_dynamic_array_spikegeneratorgroup_neuron_index.resize(213084));
        for(int i=0; i<_num__static_array__dynamic_array_spikegeneratorgroup_neuron_index; i++)
        {
            _dynamic_array_spikegeneratorgroup_neuron_index[i] = _static_array__dynamic_array_spikegeneratorgroup_neuron_index[i];
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_spikegeneratorgroup_neuron_index[0]),
                &_dynamic_array_spikegeneratorgroup_neuron_index[0],
                sizeof(_dynamic_array_spikegeneratorgroup_neuron_index[0])*_dynamic_array_spikegeneratorgroup_neuron_index.size(),
                hipMemcpyHostToDevice
            )
        );
        _dynamic_array_spikegeneratorgroup_spike_time.resize(213084);
        THRUST_CHECK_ERROR(dev_dynamic_array_spikegeneratorgroup_spike_time.resize(213084));
        for(int i=0; i<_num__static_array__dynamic_array_spikegeneratorgroup_spike_time; i++)
        {
            _dynamic_array_spikegeneratorgroup_spike_time[i] = _static_array__dynamic_array_spikegeneratorgroup_spike_time[i];
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_spikegeneratorgroup_spike_time[0]),
                &_dynamic_array_spikegeneratorgroup_spike_time[0],
                sizeof(_dynamic_array_spikegeneratorgroup_spike_time[0])*_dynamic_array_spikegeneratorgroup_spike_time.size(),
                hipMemcpyHostToDevice
            )
        );
        _dynamic_array_spikegeneratorgroup__timebins.resize(213084);
        THRUST_CHECK_ERROR(dev_dynamic_array_spikegeneratorgroup__timebins.resize(213084));
        _array_spikegeneratorgroup__lastindex[0] = 0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_spikegeneratorgroup__lastindex + 0,
                                    &_array_spikegeneratorgroup__lastindex[0],
                                    sizeof(_array_spikegeneratorgroup__lastindex[0]),
                                    hipMemcpyHostToDevice
                                )
                            );
        _array_spikegeneratorgroup_period[0] = 0.0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_spikegeneratorgroup_period + 0,
                                    &_array_spikegeneratorgroup_period[0],
                                    sizeof(_array_spikegeneratorgroup_period[0]),
                                    hipMemcpyHostToDevice
                                )
                            );
        for(int i=0; i<_num__static_array__array_neurongroup_c; i++)
        {
            _array_neurongroup_c[i] = _static_array__array_neurongroup_c[i];
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_neurongroup_c,
                &_array_neurongroup_c[0],
                sizeof(_array_neurongroup_c[0])*_num__array_neurongroup_c,
                hipMemcpyHostToDevice
            )
        );
        for(int i=0; i<_num__array_neurongroup_1_lastspike; i++)
        {
            _array_neurongroup_1_lastspike[i] = - 10000.0;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_neurongroup_1_lastspike,
                &_array_neurongroup_1_lastspike[0],
                sizeof(_array_neurongroup_1_lastspike[0])*_num__array_neurongroup_1_lastspike,
                hipMemcpyHostToDevice
            )
        );
        for(int i=0; i<_num__array_neurongroup_1_not_refractory; i++)
        {
            _array_neurongroup_1_not_refractory[i] = true;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                dev_array_neurongroup_1_not_refractory,
                &_array_neurongroup_1_not_refractory[0],
                sizeof(_array_neurongroup_1_not_refractory[0])*_num__array_neurongroup_1_not_refractory,
                hipMemcpyHostToDevice
            )
        );
        _dynamic_array_synapses_2_delay.resize(1);
        THRUST_CHECK_ERROR(dev_dynamic_array_synapses_2_delay.resize(1));
        _dynamic_array_synapses_2_delay.resize(1);
        THRUST_CHECK_ERROR(dev_dynamic_array_synapses_2_delay.resize(1));
        _dynamic_array_synapses_2_delay[0] = 0.001;
        _run_synapses_2_synapses_create_generator_codeobject();
        for(int i=0; i<_dynamic_array_synapses_2_w.size(); i++)
        {
            _dynamic_array_synapses_2_w[i] = 0.7;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_synapses_2_w[0]),
                &_dynamic_array_synapses_2_w[0],
                sizeof(_dynamic_array_synapses_2_w[0])*_dynamic_array_synapses_2_w.size(),
                hipMemcpyHostToDevice
            )
        );
        _run_synapses_3_synapses_create_generator_codeobject();
        for(int i=0; i<_dynamic_array_synapses_3_w.size(); i++)
        {
            _dynamic_array_synapses_3_w[i] = 2;
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_synapses_3_w[0]),
                &_dynamic_array_synapses_3_w[0],
                sizeof(_dynamic_array_synapses_3_w[0])*_dynamic_array_synapses_3_w.size(),
                hipMemcpyHostToDevice
            )
        );
        _array_networkoperation_clock_dt[0] = 0.0001;
        _array_networkoperation_clock_dt[0] = 0.0001;
        _array_networkoperation_1_clock_dt[0] = 10.0;
        _array_networkoperation_1_clock_dt[0] = 10.0;
        _array_networkoperation_2_clock_dt[0] = 10.0;
        _array_networkoperation_2_clock_dt[0] = 10.0;
        _array_networkoperation_1_clock_timestep[0] = 0;
        _array_networkoperation_1_clock_t[0] = 0.0;
        _array_networkoperation_2_clock_timestep[0] = 0;
        _array_networkoperation_2_clock_t[0] = 0.0;
        _array_defaultclock_timestep[0] = 0;
        _array_defaultclock_t[0] = 0.0;
        _array_networkoperation_clock_timestep[0] = 0;
        _array_networkoperation_clock_t[0] = 0.0;
        _array_spikegeneratorgroup__lastindex[0] = 0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_spikegeneratorgroup__lastindex + 0,
                                    &_array_spikegeneratorgroup__lastindex[0],
                                    sizeof(_array_spikegeneratorgroup__lastindex[0]),
                                    hipMemcpyHostToDevice
                                )
                            );
        for(int i=0; i<_num__static_array__dynamic_array_spikegeneratorgroup__timebins; i++)
        {
            _dynamic_array_spikegeneratorgroup__timebins[i] = _static_array__dynamic_array_spikegeneratorgroup__timebins[i];
        }
        CUDA_SAFE_CALL(
            hipMemcpy(
                thrust::raw_pointer_cast(&dev_dynamic_array_spikegeneratorgroup__timebins[0]),
                &_dynamic_array_spikegeneratorgroup__timebins[0],
                sizeof(_dynamic_array_spikegeneratorgroup__timebins[0])*_dynamic_array_spikegeneratorgroup__timebins.size(),
                hipMemcpyHostToDevice
            )
        );
        _array_spikegeneratorgroup__period_bins[0] = 0.0;
                            CUDA_SAFE_CALL(
                                hipMemcpy(
                                    dev_array_spikegeneratorgroup__period_bins + 0,
                                    &_array_spikegeneratorgroup__period_bins[0],
                                    sizeof(_array_spikegeneratorgroup__period_bins[0]),
                                    hipMemcpyHostToDevice
                                )
                            );
        _before_run_synapses_2_pre_push_spikes();
        _before_run_synapses_3_pre_push_spikes();
        _before_run_synapses_pre_push_spikes();
        _before_run_synapses_1_post_push_spikes();
        dev_dynamic_array_synapses_2__synaptic_pre.clear();
        dev_dynamic_array_synapses_2__synaptic_pre.shrink_to_fit();
        dev_dynamic_array_synapses_3__synaptic_pre.clear();
        dev_dynamic_array_synapses_3__synaptic_pre.shrink_to_fit();
        dev_dynamic_array_synapses__synaptic_pre.clear();
        dev_dynamic_array_synapses__synaptic_pre.shrink_to_fit();
        dev_dynamic_array_synapses_delay.clear();
        dev_dynamic_array_synapses_delay.shrink_to_fit();
        dev_dynamic_array_synapses_1_delay.clear();
        dev_dynamic_array_synapses_1_delay.shrink_to_fit();
        dev_dynamic_array_synapses_2_delay.clear();
        dev_dynamic_array_synapses_2_delay.shrink_to_fit();
        dev_dynamic_array_synapses_3_delay.clear();
        dev_dynamic_array_synapses_3_delay.shrink_to_fit();
        magicnetwork.clear();
        magicnetwork.add(&networkoperation_1_clock, _run_random_number_buffer);
        magicnetwork.add(&networkoperation_2_clock, _run_random_number_buffer);
        magicnetwork.add(&defaultclock, _run_random_number_buffer);
        magicnetwork.add(&networkoperation_clock, _run_random_number_buffer);
        magicnetwork.add(&defaultclock, _run_synapses_1_summed_variable_Igap_post_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_1_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_stateupdater_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_1_spike_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_spike_thresholder_codeobject);
        magicnetwork.add(&defaultclock, _run_spikegeneratorgroup_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_2_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_2_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_3_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_3_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_pre_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_pre_codeobject);
        magicnetwork.add(&defaultclock, _run_synapses_1_post_push_spikes);
        magicnetwork.add(&defaultclock, _run_synapses_1_post_codeobject);
        magicnetwork.add(&defaultclock, _run_neurongroup_2_spike_resetter_codeobject);
        magicnetwork.add(&networkoperation_1_clock, NULL);
        magicnetwork.add(&networkoperation_2_clock, NULL);
        magicnetwork.add(&networkoperation_clock, NULL);
        magicnetwork.run(500.0, NULL, 10.0);
        random_number_buffer.run_finished();
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        CUDA_SAFE_CALL(hipProfilerStop());
        _after_run_neurongroup_1_spike_thresholder_codeobject();
        _after_run_neurongroup_2_spike_thresholder_codeobject();
        _debugmsg_synapses_2_pre_codeobject();
        
        _debugmsg_synapses_3_pre_codeobject();
        
        _debugmsg_synapses_pre_codeobject();
        
        _debugmsg_synapses_1_post_codeobject();

    }

    //const double _run_time3 = (double)(std::clock() -_start_time3)/CLOCKS_PER_SEC;
    //printf("INFO: main_lines took %f seconds\n", _run_time3);

        

    brian_end();
        


    // Profiling
    //const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    //printf("INFO: main function took %f seconds\n", _run_time);

    return 0;
}